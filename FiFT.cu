#include "hip/hip_runtime.h"
#include "FiFT.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define BASE_BLOCK 16



FiFT::FiFT(const size_t burst_size, const size_t batch_size)
    : m_burst_size(burst_size)
    , m_batch_size(batch_size)
    , m_num_elts(batch_size * burst_size)
{
    size_t buf_size = burst_size * batch_size * sizeof(COMPLEX_T);
    checkCudaErrors(hipMalloc(&m_workspace, buf_size));
};

FiFT::~FiFT() {
    checkCudaErrors(hipFree(m_workspace));
};



/*
  TODO: One possible issue with this step1 approach is the high shared memory per thread, which 
  might reduce occupancy (and I think it might have high compute usage relative to bandwidth?).
  Could try a version which does the loads across batches but then does the matmul 
  as a cooperative warp. Would  the increased need for sync hurt that?
*/

#define STEP1_THREADBLOCK 32

__global__ static void FFT_step1(const REAL_T* input,
				 COMPLEX_T* output,
				 const size_t burst_size,
				 const size_t batch_size)
{
    const int burst =  blockDim.x * blockIdx.x + threadIdx.x;
    if (burst >= batch_size) return;

    const int num_blocks = burst_size / BASE_BLOCK;

    // TODO: store as real or complex? i.e. cast ost or shared mem cost?
    __shared__ REAL_T shared_mem[BASE_BLOCK * STEP1_THREADBLOCK];

    // Multiply each block by the base case twiddle matrix
    for (int block = 0; block < num_blocks; ++block) {

	// Read the whole block into shared mem in a loop
	// Store it transposed, so adjacent threads aren't getting bank conflicts
	const REAL_T* global_block = &input[burst + block * batch_size];
	REAL_T* local_block = &shared_mem[threadIdx.x];
	for (int k = 0; k < BASE_BLOCK; ++k) {
	    local_block[k * STEP1_THREADBLOCK] = global_block[k * num_blocks * batch_size];
	}
	
	// Each element in the output block
	for (int k = 0; k < BASE_BLOCK; ++k) {
	    COMPLEX_T y_k = {0.0f, 0.0f};

	    // Multiply the block by a row from the twiddle matrix
	    for (int n = 0; n < BASE_BLOCK; ++n) {
		// TODO: do the multiplication by hand, removing the zero terms
		COMPLEX_T term = {(float) local_block[n * STEP1_THREADBLOCK], 0};
		float exponent = -2.0 * 3.141592653589793 * n * k / (float) BASE_BLOCK;
		COMPLEX_T twiddle = {cos(exponent), sin(exponent)};
		y_k = hipCaddf(y_k, hipCmulf(term, twiddle));
	    }
	    
	    output[burst + (block + k * num_blocks) * batch_size] = y_k;
	}
    }
    
}
    

__global__ void copy_kernel(const REAL_T* input, COMPLEX_T* output, const size_t n) {
    int i =  blockDim.x * blockIdx.x + threadIdx.x;
    if (i > n) return;
    output[i] = make_hipComplex (input[i], 0.f);
}

void FiFT::run(const REAL_T* input, COMPLEX_T* output) {
    int num_blocks = (m_batch_size + STEP1_THREADBLOCK - 1) / STEP1_THREADBLOCK;
    FFT_step1<<<num_blocks, STEP1_THREADBLOCK>>>(input,
						 output,
						 m_burst_size,
						 m_batch_size);
};



// ------------------------------ VALIDATION WRAPPER ------------------------------ //

extern "C"
void test(const REAL_T* input,
	  COMPLEX_T* output,
	  const size_t burst_size,
	  const size_t batch_size)
{

    const size_t input_size = burst_size * batch_size * sizeof(REAL_T);
    const size_t output_size = burst_size * batch_size * sizeof(COMPLEX_T);
    REAL_T *d_input;
    COMPLEX_T *d_output;
    checkCudaErrors(hipMalloc((void**)&d_input, input_size));
    checkCudaErrors(hipMalloc((void**)&d_output, output_size));
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    
    FiFT fift(burst_size, batch_size);
    fift.run(d_input, d_output);

    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}