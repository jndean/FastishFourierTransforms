#include "hip/hip_runtime.h"
#include "FiFT.h"

#include <algorithm>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define PI 3.141592653589793

#define BASE_BLOCK 32
#define BASE_BLOCK_LOG2 5
#define BASE_BLOCK_MASK (BASE_BLOCK - 1)


__device__ __constant__ COMPLEX_T s1_twiddles[BASE_BLOCK][BASE_BLOCK];
__device__ __constant__ COMPLEX_T s2_twiddles[1024];


FiFT::FiFT(const size_t burst_size, const size_t batch_size)
    : m_burst_size(burst_size)
    , m_batch_size(batch_size)
    , m_num_elts(batch_size * burst_size)
{
    // Allocate workspace
    size_t buf_size = burst_size * batch_size * sizeof(COMPLEX_T);
    checkCudaErrors(hipMalloc(&m_workspace, buf_size));

    // Precompute FFT twiddles and put them in constant memory
    COMPLEX_T h_s1_twiddles[BASE_BLOCK][BASE_BLOCK];
    COMPLEX_T h_s2_twiddles[burst_size/2];
    for (int k = 0; k < burst_size/2; ++k) {
	double exponent = -2.0 * PI * k / (double)burst_size;
	h_s2_twiddles[k] = {(float) cos(exponent), (float) sin(exponent)};
    }
    for (int n = 0; n < BASE_BLOCK; ++n) {
	for (int k = 0; k < BASE_BLOCK; ++k) {
	    double exponent = -2.0 * PI * n * k / (double) BASE_BLOCK;
	    h_s1_twiddles[n][k] = {(float) cos(exponent), (float) sin(exponent)};
	}
    }
    hipMemcpyToSymbol(HIP_SYMBOL(s1_twiddles), h_s1_twiddles, sizeof(s1_twiddles));
    hipMemcpyToSymbol(HIP_SYMBOL(s2_twiddles), h_s2_twiddles, sizeof(s2_twiddles));
};

FiFT::~FiFT() {
    checkCudaErrors(hipFree(m_workspace));
};



// ----------------------------------- NORMAL VERSION ----------------------------------- //



#define STEP1_THREADBLOCK 32

__global__ static void step1_kernel(const REAL_T* input,
				    COMPLEX_T* output,
				    const size_t burst_size,
				    const size_t batch_size)
{
    const int burst =  blockDim.x * blockIdx.x + threadIdx.x;
    if (burst >= batch_size) return;

    const int num_blocks = burst_size >> BASE_BLOCK_LOG2;

    // TODO: store as real or complex? i.e. cast ost or shared mem cost?
    __shared__ REAL_T shared_mem[BASE_BLOCK * STEP1_THREADBLOCK];

    // Multiply each block by the base case twiddle matrix
    for (int block = 0; block < num_blocks; ++block) {

	// Read the whole block into shared mem in a loop
	// Store it transposed, so adjacent threads aren't getting bank conflicts
	const REAL_T* global_block = &input[burst + block * batch_size];
	REAL_T* local_block = &shared_mem[threadIdx.x];
	for (int k = 0; k < BASE_BLOCK; ++k) {
	    local_block[k * STEP1_THREADBLOCK] = global_block[k * num_blocks * batch_size];
	}
	
	// Each element in the output block
	for (int k = 0; k < BASE_BLOCK; ++k) {
	    COMPLEX_T y_k = {0.0f, 0.0f};

	    // Multiply the block by a row from the twiddle matrix
	    for (int n = 0; n < BASE_BLOCK; ++n) {
		// TODO: do the multiplication by hand, removing the zero terms
		COMPLEX_T term = {(float) local_block[n * STEP1_THREADBLOCK], 0};
		//COMPLEX_T twiddle = {0.0f, 0.0f};
		COMPLEX_T twiddle = s1_twiddles[n][k];
		y_k = hipCaddf(y_k, hipCmulf(term, twiddle));
	    }
	    
	    output[burst + (block + k * num_blocks) * batch_size] = y_k;
	}
    } 
}


void FiFT::run_step1(const REAL_T* input, COMPLEX_T* output)
{
    int num_blocks = (m_batch_size + STEP1_THREADBLOCK - 1) / STEP1_THREADBLOCK;
    step1_kernel<<<num_blocks, STEP1_THREADBLOCK>>>(input,
						    output,
						    m_burst_size,
						    m_batch_size);
}


#define STEP2_THREADBLOCK 32

__global__ static void step2_kernel(const COMPLEX_T* input,
				    COMPLEX_T* output,
				    const int block_size,
				    const int num_blocks,
				    const int batch_size,
				    const int burst_size)
{
    const int burst =  blockDim.x * blockIdx.x + threadIdx.x;
    if (burst >= batch_size) return;
    
    const int half_block_size = block_size >> 1;
    
    for (int block = 0; block < num_blocks; ++block) {

	float exponent = -2.0 * PI * block * half_block_size / (float)burst_size;
	COMPLEX_T twiddle = {cos(exponent), sin(exponent)};
	
	for (int i = 0; i < half_block_size; ++i) {
	    COMPLEX_T odd = input[(block * block_size + half_block_size + i) * batch_size + burst];
	    COMPLEX_T even = input[(block * block_size + i) * batch_size + burst];
	    odd = hipCmulf(odd, twiddle);
	    output[(block * half_block_size + i) * batch_size + burst] = hipCaddf(even, odd);
	    output[((num_blocks + block) * half_block_size + i) * batch_size + burst] = hipCsubf(even, odd);
	}
    }
} 


void FiFT::run_step2(COMPLEX_T* input, COMPLEX_T* output)
{
    int num_FFT_blocks = BASE_BLOCK;
    int FFT_block_size = m_burst_size / num_FFT_blocks;

    COMPLEX_T *read = input, *write = output;

    while (num_FFT_blocks < m_burst_size) {
	int num_thread_blocks = (m_batch_size + STEP2_THREADBLOCK - 1) / STEP2_THREADBLOCK;
	step2_kernel<<<num_thread_blocks, STEP2_THREADBLOCK>>>(read,
							       write,
							       FFT_block_size,
							       num_FFT_blocks,
							       m_batch_size,
							       m_burst_size);
	std::swap(read, write);
	num_FFT_blocks *= 2;
	FFT_block_size /= 2;
    }
}


void FiFT::run(const REAL_T* input, COMPLEX_T* output) {
    COMPLEX_T *step2_input = m_workspace;
    COMPLEX_T *step2_output = output;
    int N = m_burst_size >> BASE_BLOCK_LOG2, log2N = 1;
    while (N >>= 1) log2N++;
    if (log2N & 1) std::swap(step2_input, step2_output);
    
    run_step1(input, step2_input);
    run_step2(step2_input, step2_output);
};



// ----------------------------------- TRANSPOSED VERSION ----------------------------------- //



__global__ static void step1_transpose_kernel(const REAL_T* input,
					      COMPLEX_T* output,
					      const size_t burst_size,
					      const size_t batch_size)
{
    const int burst =  blockDim.x * blockIdx.x + threadIdx.x;
    if (burst >= batch_size) return;

    const int num_blocks = burst_size >> BASE_BLOCK_LOG2;

    // Store as float not int8. Saves on later casts, and means there's one
    // element per shared memory bank
    __shared__ float tile[BASE_BLOCK * BASE_BLOCK];

    for (int block = 0; block < num_blocks; ++block) {

	// Read the whole block into shared memory
	// Store it transposed, so adjacent threads aren't getting bank conflicts
	const REAL_T* global_block = &input[burst + block * batch_size];
	float* local_block = &tile[threadIdx.x];
	for (int k = 0; k < BASE_BLOCK; ++k) {
	    local_block[k * BASE_BLOCK] = global_block[k * num_blocks * batch_size];
	}
	
	// Each element in the output block
	for (int k = 0; k < BASE_BLOCK; ++k) {
	    COMPLEX_T y_k = {0.0f, 0.0f};

	    // Multiply the block by a row from the twiddle matrix
	    #pragma unroll
	    for (int n = 0; n < BASE_BLOCK; ++n) {
		// Seems it's still faster to do complex multiplication with a zero complex term than
		// do the mul separately with fewer terms (does it compile to use double instrucitons maybe?)
		COMPLEX_T term = {(float)local_block[n * BASE_BLOCK], 0};
		
		// Twiddle matrix is symmetric, so can just put [k][n] the way round that means
		// __constant__ memory access is done with broadcasts :D
		COMPLEX_T twiddle = s1_twiddles[k][n];
		y_k = hipCaddf(y_k, hipCmulf(term, twiddle));
	    }
	    
	    //output[burst + (block + k * num_blocks) * batch_size] = y_k;
	    // These writes aren't coalesced, and it's a big deal...
	    output[burst * burst_size + k + block * BASE_BLOCK] = y_k;
	}
    }

}


void FiFT::run_step1_transpose(const REAL_T* input, COMPLEX_T* output)
{
    int num_blocks = (m_batch_size + BASE_BLOCK - 1) >> BASE_BLOCK_LOG2;
    step1_transpose_kernel<<<num_blocks, BASE_BLOCK>>>(input,
						       output,
						       m_burst_size,
						       m_batch_size);
}


__global__ static void step2_transpose_kernel(const COMPLEX_T* input,
					      COMPLEX_T* output,
					      const int burst_size)
{
    const int burst = blockIdx.x;
    const int element = threadIdx.x;

    extern __shared__ COMPLEX_T local[];
    // Load elements accounting for how they're twisted within the burst by step1
    int num_base_blocks = burst_size >> BASE_BLOCK_LOG2;
    int step1_block = element >> BASE_BLOCK_LOG2;
    int step1_block_element = element & BASE_BLOCK_MASK;
    int local_idx = step1_block + step1_block_element * num_base_blocks;
    local[local_idx] = input[burst * burst_size + element];
    local_idx += num_base_blocks >> 1;
    local[local_idx] = input[burst * burst_size + element + burst_size/2];

    int num_blocks = BASE_BLOCK;
    int block_size = num_base_blocks;
    int half_block_size = block_size >> 1;

    while (num_blocks < burst_size) {
	
	int block = element / half_block_size;
	int block_elt = element % half_block_size;
	
	// This is where I might try an async shared mem load? If my GPU supported it...
	int idx = block * block_size + block_elt;
	COMPLEX_T even = local[idx];
	COMPLEX_T odd = local[idx + half_block_size];

	idx = block * half_block_size;
	COMPLEX_T twiddle = s2_twiddles[idx];
	odd = hipCmulf(odd, twiddle);
	idx += block_elt;
	
	__syncthreads();
	
	local[idx] = hipCaddf(even, odd);
	local[idx + burst_size/2] = hipCsubf(even, odd);
	
	num_blocks <<= 1;
	block_size = half_block_size;
	half_block_size >>= 1;	
    }

    __syncthreads();
    
    output[burst * burst_size + element] = local[element];
    output[burst * burst_size + element + burst_size/2] = local[element + burst_size/2];
} 


void FiFT::run_step2_transpose(COMPLEX_T* input, COMPLEX_T* output)
{
    const int num_blocks = m_batch_size;
    const int threads_per_block = m_burst_size / 2;
    const int sharedmem = sizeof(COMPLEX_T) * m_burst_size;

    step2_transpose_kernel<<<num_blocks, threads_per_block, sharedmem>>>
	(input,
	 output,
	 m_burst_size);
}


void FiFT::run_transposed(const REAL_T* input, COMPLEX_T* output) {
    run_step1_transpose(input, m_workspace);
    run_step2_transpose(m_workspace, output);
};


// ----------------------------------- PACKED VERSION ----------------------------------- //


__global__ static void step1_packed_kernel(const COMPLEX_T* input,
					   COMPLEX_T* output,
					   const size_t burst_size,
					   const size_t batch_size)
{
    const int burst =  blockDim.x * blockIdx.x + threadIdx.x;
    if (burst >= batch_size) return;

    const int num_blocks = burst_size >> BASE_BLOCK_LOG2;

    // Store as float not int8. Saves on later casts, and means there's one
    // element per shared memory bank
    __shared__ COMPLEX_T tile[BASE_BLOCK * BASE_BLOCK];

    for (int block = 0; block < num_blocks; ++block) {

	// Read the whole block into shared memory
	// Store it transposed, so adjacent threads aren't getting bank conflicts
	const COMPLEX_T* global_block = &input[burst + block * batch_size];
	// TODO: Now that this is COMPLEX_T, get 2-way bank conflicts on every access...
	// Could split into real components in a column to avoid this
	COMPLEX_T* local_block = &tile[threadIdx.x];
	for (int k = 0; k < BASE_BLOCK; ++k) {
	    local_block[k * BASE_BLOCK] = global_block[k * num_blocks * batch_size];
	}
	
	// Each element in the output block
	for (int k = 0; k < BASE_BLOCK; ++k) {
	    COMPLEX_T y_k = {0.0f, 0.0f};

	    // Multiply the block by a row from the twiddle matrix
	    // #pragma unroll
	    for (int n = 0; n < BASE_BLOCK; ++n) {
		y_k = hipCaddf(y_k, hipCmulf(s1_twiddles[k][n], local_block[n * BASE_BLOCK]));
	    }
	    
	    //output[burst + (block + k * num_blocks) * batch_size] = y_k;
	    // These writes aren't coalesced, and it's a big deal...
	    output[burst * burst_size + k + block * BASE_BLOCK] = y_k;
	}
    }

}


void FiFT::run_step1_packed(const REAL_T* input, COMPLEX_T* output)
{
    int num_blocks = (m_batch_size + BASE_BLOCK - 1) >> BASE_BLOCK_LOG2;
    step1_packed_kernel<<<num_blocks, BASE_BLOCK>>>
	((COMPLEX_T*) input,
	 output,
	 m_burst_size / 2,
	 m_batch_size);
}


__global__ static void step2_packed_kernel(const COMPLEX_T* input,
					   COMPLEX_T* output,
					   const int burst_size)
{
    const int burst = blockIdx.x;
    const int element = threadIdx.x;

    extern __shared__ COMPLEX_T local[];
    // Load elements accounting for how they're twisted within the burst by step1
    int num_base_blocks = burst_size >> BASE_BLOCK_LOG2;
    int step1_block = element >> BASE_BLOCK_LOG2;
    int step1_block_element = element & BASE_BLOCK_MASK;
    int local_idx = step1_block + step1_block_element * num_base_blocks;
    local[local_idx] = input[burst * burst_size + element];
    local_idx += num_base_blocks >> 1;
    local[local_idx] = input[burst * burst_size + element + burst_size/2];

    int num_blocks = BASE_BLOCK;
    int block_size = num_base_blocks;
    int half_block_size = block_size >> 1;

    while (num_blocks < burst_size) {
	
	int block = element / half_block_size;
	int block_elt = element % half_block_size;
	
	// This is where I might try an async shared mem load? If my GPU supported it...
	int idx = block * block_size + block_elt;
	COMPLEX_T even = local[idx];
	COMPLEX_T odd = local[idx + half_block_size];

	idx = block * half_block_size;
	COMPLEX_T twiddle = s2_twiddles[idx];
	odd = hipCmulf(odd, twiddle);
	idx += block_elt;
	
	__syncthreads();
	
	local[idx] = hipCaddf(even, odd);
	local[idx + burst_size/2] = hipCsubf(even, odd);
	
	num_blocks <<= 1;
	block_size = half_block_size;
	half_block_size >>= 1;	
    }

    __syncthreads();

    // W^nk_N = e^(-i.2.PI.n.k/N)
    // A(k) = (1 - j.WK2n) / 2
    // B(k) = (1 + j.WK2n) / 2
    int k = element;
    int Nmink = (burst_size - k) % burst_size;
    COMPLEX_T XNmink_conj = {local[Nmink].x, -local[Nmink].y};
    float exponent = -2 * PI * k / (float) burst_size;
    COMPLEX_T jWk2N = hipCmulf({0, 1}, {cos(exponent), sin(exponent)});
    COMPLEX_T Ak = hipCmulf({0.5, 0}, hipCsubf({1, 0}, jWk2N));
    COMPLEX_T Bk = hipCmulf({0.5, 0}, hipCaddf({1, 0}, jWk2N));
    COMPLEX_T Gk = hipCmulf(local[k], Ak);
    Gk = hipCaddf(Gk, hipCmulf(XNmink_conj, Bk));  
    output[burst * burst_size + element] = Gk;

    k = element + burst_size/2;
    Nmink = (burst_size - k) % burst_size;
    XNmink_conj = {local[Nmink].x, -local[Nmink].y};
    exponent = -2 * PI * k / (float) burst_size;
    jWk2N = hipCmulf({0, 1}, {cos(exponent), sin(exponent)});
    Ak = hipCmulf({0.5, 0}, hipCsubf({1, 0}, jWk2N));
    Bk = hipCmulf({0.5, 0}, hipCaddf({1, 0}, jWk2N));
    Gk = hipCmulf(local[k], Ak);
    Gk = hipCaddf(Gk, hipCmulf(XNmink_conj, Bk));  
    output[burst * burst_size + element + burst_size/2] = Gk;
} 


void FiFT::run_step2_packed(COMPLEX_T* input, COMPLEX_T* output)
{
    const int num_blocks = m_batch_size;
    const int threads_per_block = m_burst_size / 4;
    const int sharedmem = sizeof(COMPLEX_T) * m_burst_size / 2;

    step2_packed_kernel<<<num_blocks, threads_per_block, sharedmem>>>
	(input,
	 output,
	 m_burst_size/2);
}


void FiFT::run_packed(const REAL_T* input, COMPLEX_T* output) {
    run_step1_packed(input, m_workspace);
    run_step2_packed(m_workspace, output);
};



// ------------------------------ ONESHOT VERSION ------------------------------ //



#define WARP 32

__global__ static void oneshot_kernel(const REAL_T* input,
				      COMPLEX_T* output,
				      const int burst_size)
{
    
}


void FiFT::run_oneshot(const REAL_T* input, COMPLEX_T* output) {
    const int thread_block = WARP;
    const int thread_grid = (m_burst_size + WARP - 1) / WARP;
    const int shmem = m_burst_size * (sizeof(COMPLEX_T) + WARP * sizeof(REAL_T));
    oneshot_kernel<<<thread_grid, thread_block, shmem>>>
	(input, output, m_burst_size);
};



// ------------------------------ VALIDATION WRAPPERS ------------------------------ //

extern "C"
const int base_block = BASE_BLOCK;

extern "C"
void test_step1(const REAL_T* input,
		COMPLEX_T* output,
		const size_t burst_size,
		const size_t batch_size)
{

    const size_t input_size = burst_size * batch_size * sizeof(REAL_T);
    const size_t output_size = burst_size * batch_size * sizeof(COMPLEX_T);
    REAL_T *d_input;
    COMPLEX_T *d_output;
    checkCudaErrors(hipMalloc((void**)&d_input, input_size));
    checkCudaErrors(hipMalloc((void**)&d_output, output_size));
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    
    FiFT fift(burst_size, batch_size);
    fift.run_step1(d_input, d_output);

    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}


extern "C"
void test_run(const REAL_T* input,
	      COMPLEX_T* output,
	      const size_t burst_size,
	      const size_t batch_size)
{

    const size_t input_size = burst_size * batch_size * sizeof(REAL_T);
    const size_t output_size = burst_size * batch_size * sizeof(COMPLEX_T);
    REAL_T *d_input;
    COMPLEX_T *d_output;
    checkCudaErrors(hipMalloc((void**)&d_input, input_size));
    checkCudaErrors(hipMalloc((void**)&d_output, output_size));
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    
    FiFT fift(burst_size, batch_size);
    fift.run(d_input, d_output);

    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}



extern "C"
void test_step1_transpose(const REAL_T* input,
			  COMPLEX_T* output,
			  const size_t burst_size,
			  const size_t batch_size)
{
    const size_t input_size = burst_size * batch_size * sizeof(REAL_T);
    const size_t output_size = burst_size * batch_size * sizeof(COMPLEX_T);
    REAL_T *d_input;
    COMPLEX_T *d_output;
    checkCudaErrors(hipMalloc((void**)&d_input, input_size));
    checkCudaErrors(hipMalloc((void**)&d_output, output_size));
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    
    FiFT fift(burst_size, batch_size);
    fift.run_step1_transpose(d_input, d_output);

    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}


extern "C"
void test_step2_transpose(COMPLEX_T* input,
			  COMPLEX_T* output,
			  const size_t burst_size,
			  const size_t batch_size)
{

    const size_t input_size = burst_size * batch_size * sizeof(COMPLEX_T);
    const size_t output_size = burst_size * batch_size * sizeof(COMPLEX_T);
    COMPLEX_T *d_input;
    COMPLEX_T *d_output;
    checkCudaErrors(hipMalloc((void**)&d_input, input_size));
    checkCudaErrors(hipMalloc((void**)&d_output, output_size));
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));
    
    FiFT fift(burst_size, batch_size);
    fift.run_step2_transpose(d_input, d_output);
    
    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}




extern "C"
void test_step1_packed(const REAL_T* input,
			  COMPLEX_T* output,
			  const size_t burst_size,
			  const size_t batch_size)
{
    const size_t input_size = burst_size * batch_size * sizeof(REAL_T);
    const size_t output_size = burst_size * batch_size * sizeof(COMPLEX_T);
    REAL_T *d_input;
    COMPLEX_T *d_output;
    checkCudaErrors(hipMalloc((void**)&d_input, input_size));
    checkCudaErrors(hipMalloc((void**)&d_output, output_size));
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    
    FiFT fift(burst_size, batch_size);
    fift.run_step1_packed(d_input, d_output);

    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}


extern "C"
void test_step2_packed(COMPLEX_T* input,
		       COMPLEX_T* output,
		       const size_t burst_size,
		       const size_t batch_size)
{

    const size_t input_size = burst_size * batch_size * sizeof(COMPLEX_T);
    const size_t output_size = burst_size * batch_size * sizeof(COMPLEX_T);
    COMPLEX_T *d_input;
    COMPLEX_T *d_output;
    checkCudaErrors(hipMalloc((void**)&d_input, input_size));
    checkCudaErrors(hipMalloc((void**)&d_output, output_size));
    checkCudaErrors(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));
    
    FiFT fift(burst_size, batch_size);
    fift.run_step2_packed(d_input, d_output);
    
    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
}


